#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void add(int a, int b, int *sum) {
  *sum = a + b;
}

int main() {
  int *result;
  hipMalloc((void**)&result, sizeof(int));
  add<<<1, 1>>>(100, 200, result);
  hipDeviceSynchronize();

  int h_result = 0;
  hipMemcpy(&h_result, result, sizeof(int), hipMemcpyDeviceToHost);
  printf("100 + 200 = %d\n", h_result);
  return 0;
}