#include "common.h"

static const int SIZE = (64 * 1024 * 1024);

float cuda_malloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsed_time;
  const int bytes = size * sizeof(int);

  CHECK(hipEventCreate(&start));
  CHECK(hipEventCreate(&stop));

  a = new int[size];
  CHECK(hipMalloc(&dev_a, bytes));

  hipEventRecord(start, 0);
  for (int i = 0; i < 100; i++) {
    if (up) {
      CHECK(hipMemcpy(dev_a, a, bytes, hipMemcpyHostToDevice));
    } else {
      CHECK(hipMemcpy(a, dev_a, bytes, hipMemcpyDeviceToHost));
    }
  }

  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));
  CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

  delete[] a;
  CHECK(hipFree(dev_a));
  CHECK(hipEventDestroy(start));
  CHECK(hipEventDestroy(stop));

  return elapsed_time;
}

float cuda_host_malloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsed_time;
  const int bytes = size * sizeof(int);

  CHECK(hipEventCreate(&start));
  CHECK(hipEventCreate(&stop));

  CHECK(hipHostAlloc(&a, bytes, hipHostMallocDefault));
  CHECK(hipMalloc(&dev_a, bytes));

  hipEventRecord(start, 0);
  for (int i = 0; i < 100; i++) {
    if (up) {
      CHECK(hipMemcpy(dev_a, a, bytes, hipMemcpyHostToDevice));
    } else {
      CHECK(hipMemcpy(a, dev_a, bytes, hipMemcpyDeviceToHost));
    }
  }

  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));
  CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

  CHECK(hipHostFree(a));
  CHECK(hipFree(dev_a));
  CHECK(hipEventDestroy(start));
  CHECK(hipEventDestroy(stop));

  return elapsed_time;
}

int main() {
  float elapsed_time;
  float MB = float(100) * SIZE * sizeof(int) / 1024.0f / 1024.0f;

  elapsed_time = cuda_malloc_test(SIZE, true);
  printf("Time using hipMalloc:  %3.1f ms\n", elapsed_time);
  printf("\t MB/s during copy up:  %3.1f\n", MB / (elapsed_time / 1000.f));

  elapsed_time = cuda_malloc_test(SIZE, false);
  printf("Time using hipMalloc:  %3.1f ms\n", elapsed_time);
  printf("\t MB/s during copy down:  %3.1f\n", MB / (elapsed_time / 1000.f));

  elapsed_time = cuda_host_malloc_test(SIZE, true);
  printf("Time using cudaHostMalloc:  %3.1f ms\n", elapsed_time);
  printf("\t MB/s during copy up:  %3.1f\n", MB / (elapsed_time / 1000.f));

  elapsed_time = cuda_host_malloc_test(SIZE, true);
  printf("Time using cudaHostMalloc:  %3.1f ms\n", elapsed_time);
  printf("\t MB/s during copy down:  %3.1f\n", MB / (elapsed_time / 1000.f));
  return 0;
}
