#include "hip/hip_runtime.h"
#include "common.h"
#include "cpu_anim.h"
#include <vector>

static constexpr int DIM = 1024;
static constexpr float MAX_TEMP = 1.0f;
static constexpr float MIN_TEMP = 0.0001f;
static constexpr float SPEED = 0.25f;

texture<float, 2> tex_const;
texture<float, 2> tex_in;
texture<float, 2> tex_out;

__global__ void copy_const_kernel(float *iptr) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  float c = tex2D(tex_const, x, y);
  if (c != 0.0f) {
    iptr[offset] = c;
  }
}

__global__ void blend_kernel(float *out, bool dst_out) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  float t, l, c, r, b;
  if (dst_out) {
    t = tex2D(tex_in, x, y - 1);
    l = tex2D(tex_in, x - 1, y);
    c = tex2D(tex_in, x, y);
    r = tex2D(tex_in, x + 1, y);
    b = tex2D(tex_in, x, y + 1);
  } else {
    t = tex2D(tex_out, x, y - 1);
    l = tex2D(tex_out, x - 1, y);
    c = tex2D(tex_out, x, y);
    r = tex2D(tex_out, x + 1, y);
    b = tex2D(tex_out, x, y + 1);
  }
  out[offset] = c + SPEED * (t + b + r + l - 4.0f * c);
}

struct DataBlock {
  unsigned char *output_bitmap;
  float *dev_in;
  float *dev_out;
  float *dev_const;

  CPUAnimBitmap *bitmap;

  hipEvent_t start, stop;
  float total_time = .0f;
  float frames = .0f;

  void init(CPUAnimBitmap *bm) {
    bitmap = bm;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    const long image_size = bitmap->image_size();
    CHECK(hipMalloc(&output_bitmap, image_size));
    CHECK(hipMalloc(&dev_in, image_size));
    CHECK(hipMalloc(&dev_out, image_size));
    CHECK(hipMalloc(&dev_const, image_size));

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    CHECK(hipBindTexture2D(nullptr, tex_const, dev_const, desc, DIM, DIM,
                            sizeof(float) * DIM));
    CHECK(hipBindTexture2D(nullptr, tex_in, dev_in, desc, DIM, DIM,
                            sizeof(float) * DIM));
    CHECK(hipBindTexture2D(nullptr, tex_out, dev_out, desc, DIM, DIM,
                            sizeof(float) * DIM));

    std::vector<float> temp(DIM * DIM);
    for (int i = 0; i < DIM * DIM; i++) {
      temp[i] = 0;
      int x = i % DIM;
      int y = i / DIM;
      if ((x > 300) && (x < 600) && (y > 310) && (y < 601)) {
        temp[i] = MAX_TEMP;
      }
    }

    temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2.0f;
    temp[DIM * 700 + 100] = MIN_TEMP;
    temp[DIM * 300 + 300] = MIN_TEMP;
    temp[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++) {
      for (int x = 400; x < 500; x++) {
        temp[x + y * DIM] = MIN_TEMP;
      }
    }

    CHECK(hipMemcpy(dev_const, temp.data(), bitmap->image_size(),
                     hipMemcpyHostToDevice));
    for (int y = 800; y < DIM; y++) {
      for (int x = 0; x < 200; x++) {
        temp[x + y * DIM] = MAX_TEMP;
      }
    }
    CHECK(hipMemcpy(dev_in, temp.data(), bitmap->image_size(),
                     hipMemcpyHostToDevice));
  }

  ~DataBlock() {
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    CHECK(hipFree(dev_in));
    CHECK(hipFree(dev_out));
    CHECK(hipFree(dev_const));
  }
};

void anim_gpu(void *p, int) {
  auto d = static_cast<DataBlock *>(p);
  CHECK(hipEventRecord(d->start, 0));
  dim3 blocks(DIM / 16, DIM / 16);
  dim3 threads(16, 16);
  CPUAnimBitmap *bitmap = d->bitmap;

  volatile bool dst_out = true;
  for (int i = 0; i < 90; i++) {
    float *in, *out;
    if (dst_out) {
      in = d->dev_in;
      out = d->dev_out;
    } else {
      in = d->dev_out;
      out = d->dev_in;
    }
    copy_const_kernel<<<blocks, threads>>>(in);
    blend_kernel<<<blocks, threads>>>(out, dst_out);
    dst_out = !dst_out;
  }
  float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_in);
  CHECK(hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(),
                   hipMemcpyDeviceToHost));
  CHECK(hipEventRecord(d->stop, 0));
  CHECK(hipEventSynchronize(d->stop));
  float elapsed_time;
  CHECK(hipEventElapsedTime(&elapsed_time, d->start, d->stop));

  d->total_time += elapsed_time;
  ++d->frames;
  printf("Average Time per frame: %3.1f ms\n", d->total_time / d->frames);
}

void anim_exit(void *) {
  CHECK(hipUnbindTexture(tex_in));
  CHECK(hipUnbindTexture(tex_out));
  CHECK(hipUnbindTexture(tex_const));
}

int main() {
  DataBlock data;
  CPUAnimBitmap bitmap(DIM, DIM, &data);
  data.init(&bitmap);
  bitmap.anim_and_exit(anim_gpu, anim_exit);
  return 0;
}
