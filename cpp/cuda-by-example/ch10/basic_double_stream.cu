#include "hip/hip_runtime.h"
#include "common.h"

static constexpr int N = 1024 * 1024;
static constexpr int FULL_DATA_SIZE = N * 20;

__global__ void kernel(int *a, int *b, int *c) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N) {
    int idx1 = (idx + 1) % 256;
    int idx2 = (idx + 2) % 256;
    float as = (float)(a[idx] + a[idx1] + a[idx2]) / 3.0f;
    float bs = (float)(b[idx] + b[idx1] + b[idx2]) / 3.0f;
    c[idx] = int((as + bs) / 2.0f);
  }
}

int main() {
  hipDeviceProp_t prop;
  int which_device;
  CHECK(hipGetDevice(&which_device));
  CHECK(hipGetDeviceProperties(&prop, which_device));
  if (!prop.deviceOverlap) {
    printf("Device will not handle overlaps, so no "
           "speed up from streams\n");
    return 0;
  }
  hipEvent_t start, stop;
  float elapsed_time;

  CHECK(hipEventCreate(&start));
  CHECK(hipEventCreate(&stop));

  hipStream_t stream0, stream1;
  CHECK(hipStreamCreate(&stream0));
  CHECK(hipStreamCreate(&stream1));

  int *host_a, *host_b, *host_c;
  int *dev_a0, *dev_b0, *dev_c0;
  int *dev_a1, *dev_b1, *dev_c1;

  CHECK(hipMalloc(&dev_a0, N * sizeof(int)));
  CHECK(hipMalloc(&dev_b0, N * sizeof(int)));
  CHECK(hipMalloc(&dev_c0, N * sizeof(int)));
  CHECK(hipMalloc(&dev_a1, N * sizeof(int)));
  CHECK(hipMalloc(&dev_b1, N * sizeof(int)));
  CHECK(hipMalloc(&dev_c1, N * sizeof(int)));

  CHECK(hipHostAlloc(&host_a, FULL_DATA_SIZE * sizeof(int),
                      hipHostMallocDefault));
  CHECK(hipHostAlloc(&host_b, FULL_DATA_SIZE * sizeof(int),
                      hipHostMallocDefault));
  CHECK(hipHostAlloc(&host_c, FULL_DATA_SIZE * sizeof(int),
                      hipHostMallocDefault));

  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    host_a[i] = rand();
    host_b[i] = rand();
  }

  CHECK(hipEventRecord(start, 0));
  for (int i = 0; i < FULL_DATA_SIZE; i += N * 2) {
    CHECK(hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int),
                          hipMemcpyHostToDevice, stream0));
    CHECK(hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int),
                          hipMemcpyHostToDevice, stream0));

    CHECK(hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int),
                          hipMemcpyHostToDevice, stream1));
    CHECK(hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int),
                          hipMemcpyHostToDevice, stream1));


    kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
    kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

    CHECK(hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int),
                          hipMemcpyDeviceToHost, stream0));
    CHECK(hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int),
                          hipMemcpyDeviceToHost, stream1));
  }

  CHECK(hipStreamSynchronize(stream0));
  CHECK(hipStreamSynchronize(stream1));
  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));
  CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

  printf("Time taken:  %3.1f ms\n", elapsed_time);

  CHECK(hipHostFree(host_a));
  CHECK(hipHostFree(host_b));
  CHECK(hipHostFree(host_c));
  CHECK(hipFree(dev_a0));
  CHECK(hipFree(dev_b0));
  CHECK(hipFree(dev_c0));
  CHECK(hipFree(dev_a1));
  CHECK(hipFree(dev_b1));
  CHECK(hipFree(dev_c1));
  CHECK(hipEventDestroy(start));
  CHECK(hipEventDestroy(stop));
  CHECK(hipStreamDestroy(stream0));
  CHECK(hipStreamDestroy(stream1));
  return 0;
}
