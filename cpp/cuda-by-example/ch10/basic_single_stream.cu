#include "hip/hip_runtime.h"
#include "common.h"

static constexpr int N = 1024 * 1024;
static constexpr int FULL_DATA_SIZE = N * 20;

__global__ void kernel(int *a, int *b, int *c) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N) {
    int idx1 = (idx + 1) % 256;
    int idx2 = (idx + 2) % 256;
    float as = (float)(a[idx] + a[idx1] + a[idx2]) / 3.0f;
    float bs = (float)(b[idx] + b[idx1] + b[idx2]) / 3.0f;
    c[idx] = int((as + bs) / 2.0f);
  }
}

int main() {
  hipDeviceProp_t prop;
  int which_device;
  CHECK(hipGetDevice(&which_device));
  CHECK(hipGetDeviceProperties(&prop, which_device));
  if (!prop.deviceOverlap) {
    printf("Device will not handle overlaps, so no "
           "speed up from streams\n");
    return 0;
  }
  hipEvent_t start, stop;
  float elapsed_time;

  CHECK(hipEventCreate(&start));
  CHECK(hipEventCreate(&stop));
  hipStream_t stream;
  CHECK(hipStreamCreate(&stream));

  int *host_a, *host_b, *host_c;
  int *dev_a, *dev_b, *dev_c;

  CHECK(hipMalloc(&dev_a, N * sizeof(int)));
  CHECK(hipMalloc(&dev_b, N * sizeof(int)));
  CHECK(hipMalloc(&dev_c, N * sizeof(int)));

  CHECK(hipHostAlloc(&host_a, FULL_DATA_SIZE * sizeof(int),
                      hipHostMallocDefault));
  CHECK(hipHostAlloc(&host_b, FULL_DATA_SIZE * sizeof(int),
                      hipHostMallocDefault));
  CHECK(hipHostAlloc(&host_c, FULL_DATA_SIZE * sizeof(int),
                      hipHostMallocDefault));

  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    host_a[i] = rand();
    host_b[i] = rand();
  }

  CHECK(hipEventRecord(start, 0));
  for (int i = 0; i < FULL_DATA_SIZE; i += N) {
    CHECK(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int),
                          hipMemcpyHostToDevice, stream));
    CHECK(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int),
                          hipMemcpyHostToDevice, stream));

    kernel<<<N / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

    CHECK(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int),
                          hipMemcpyDeviceToHost, stream));
  }

  CHECK(hipStreamSynchronize(stream));
  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));
  CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

  printf("Time taken:  %3.1f ms\n", elapsed_time);

  CHECK(hipHostFree(host_a));
  CHECK(hipHostFree(host_b));
  CHECK(hipHostFree(host_c));
  CHECK(hipFree(dev_a));
  CHECK(hipFree(dev_b));
  CHECK(hipFree(dev_c));
  CHECK(hipEventDestroy(start));
  CHECK(hipEventDestroy(stop));
  CHECK(hipStreamDestroy(stream));
  return 0;
}
